#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <GL/glew.h>
#include <GLFW/glfw3.h>

const int screenWidth = 1920;
const int screenHeight = 1080;
const int numParticles = 10000;
const int numColors = 5;

struct Particle
{
    float3 position;
    float3 velocity;
    int color;
};

__global__ void initParticles(Particle *particles, unsigned int seed)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numParticles)
    {
        hiprandState state;
        hiprand_init(seed, idx, 0, &state);

        particles[idx].position = make_float3(hiprand_uniform(&state) * 2.0f - 1.0f, hiprand_uniform(&state) * 2.0f - 1.0f, hiprand_uniform(&state) * 2.0f - 1.0f);
        particles[idx].velocity = make_float3(0.0f, 0.0f, 0.0f);
        particles[idx].color = idx % numColors;
    }
}

__global__ void initColors(float4 *colors, unsigned int seed)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numColors)
    {
        hiprandState state;
        hiprand_init(seed, idx, 0, &state);

        colors[idx] = make_float4(hiprand_uniform(&state), hiprand_uniform(&state), hiprand_uniform(&state), 1.0f);
    }
}

__device__ float length(float3 v)
{
    return sqrtf(v.x * v.x + v.y * v.y + v.z * v.z);
}

__device__ float3 normalize(float3 v)
{
    float len = length(v);
    if (len > 0)
    {
        v.x /= len;
        v.y /= len;
        v.z /= len;
    }
    return v;
}

__device__ float calculateForce(float distance, float attractionFactor, float beta)
{
    if (distance < beta)
    {
        return (distance / beta) - 1;
    }
    else if (distance < 1.0f)
    {
        return attractionFactor * (1 - (2 * distance - 1 - beta) / (1 - beta));
    }
    else
    {
        return 0.0f;
    }
}

__global__ void updateParticles(Particle *particles, float *attractionMatrix, int numParticles, float deltaTime, int numColors, float beta, float frictionFactor)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numParticles)
    {
        float3 force = make_float3(0, 0, 0);
        Particle &self = particles[idx];

        for (int j = 0; j < numParticles; j++)
        {
            if (j != idx)
            {
                Particle &other = particles[j];
                float3 diff = make_float3(other.position.x - self.position.x, other.position.y - self.position.y, other.position.z - self.position.z);
                float distance = length(diff);

                if (distance < 1.0f)
                {
                    int color_i = self.color;
                    int color_j = other.color;
                    float attractionFactor = attractionMatrix[color_i * numColors + color_j];

                    float forceMagnitude = calculateForce(distance, attractionFactor, beta);
                    force = make_float3(force.x + forceMagnitude * normalize(diff).x, force.y + forceMagnitude * normalize(diff).y, force.z + forceMagnitude * normalize(diff).z);
                }
            }
        }

        self.velocity = make_float3(self.velocity.x * frictionFactor, self.velocity.y * frictionFactor, self.velocity.z * frictionFactor);

        float3 acceleration = force;
        self.velocity = make_float3(self.velocity.x + acceleration.x * deltaTime, self.velocity.y + acceleration.y * deltaTime, self.velocity.z + acceleration.z * deltaTime);
        self.position = make_float3(self.position.x + self.velocity.x * deltaTime, self.position.y + self.velocity.y * deltaTime, self.position.z + self.velocity.z * deltaTime);

        if (self.position.x > 1.0f)
            self.position.x = -1.0f;
        if (self.position.x < -1.0f)
            self.position.x = 1.0f;
        if (self.position.y > 1.0f)
            self.position.y = -1.0f;
        if (self.position.y < -1.0f)
            self.position.y = 1.0f;
        if (self.position.z > 1.0f)
            self.position.z = -1.0f;
        if (self.position.z < -1.0f)
            self.position.z = 1.0f;
    }
}

const char *vertexShaderSource = "#version 330 core\n"
                                 "layout (location = 0) in vec3 aPos;\n"
                                 "layout (location = 1) in int aColorIndex;\n"
                                 "out vec4 ourColor;\n"
                                 "uniform vec4 colors[3];\n"
                                 "void main()\n"
                                 "{\n"
                                 "   gl_Position = vec4(aPos, 1.0);\n"
                                 "   ourColor = colors[aColorIndex];\n"
                                 "}\0";

const char *fragmentShaderSource = "#version 330 core\n"
                                   "in vec4 ourColor;\n"
                                   "out vec4 FragColor;\n"
                                   "void main()\n"
                                   "{\n"
                                   "   FragColor = ourColor;\n"
                                   "}\n\0";

int main()
{
    if (!glfwInit())
    {
        std::cerr << "Failed to initialize GLFW" << std::endl;
        return -1;
    }

    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
    glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);

    GLFWwindow *window = glfwCreateWindow(800, 600, "Particle Simulation", NULL, NULL);
    if (window == NULL)
    {
        std::cerr << "Failed to create GLFW window" << std::endl;
        glfwTerminate();
        return -1;
    }
    glfwMakeContextCurrent(window);

    if (glewInit() != GLEW_OK)
    {
        std::cerr << "Failed to initialize GLEW" << std::endl;
        return -1;
    }

    glViewport(0, 0, 800, 600);

    unsigned int vertexShader = glCreateShader(GL_VERTEX_SHADER);
    glShaderSource(vertexShader, 1, &vertexShaderSource, NULL);
    glCompileShader(vertexShader);

    int success;
    char infoLog[512];
    glGetShaderiv(vertexShader, GL_COMPILE_STATUS, &success);
    if (!success)
    {
        glGetShaderInfoLog(vertexShader, 512, NULL, infoLog);
        std::cerr << "ERROR::SHADER::VERTEX::COMPILATION_FAILED\n"
                  << infoLog << std::endl;
    }

    unsigned int fragmentShader = glCreateShader(GL_FRAGMENT_SHADER);
    glShaderSource(fragmentShader, 1, &fragmentShaderSource, NULL);
    glCompileShader(fragmentShader);

    glGetShaderiv(fragmentShader, GL_COMPILE_STATUS, &success);
    if (!success)
    {
        glGetShaderInfoLog(fragmentShader, 512, NULL, infoLog);
        std::cerr << "ERROR::SHADER::FRAGMENT::COMPILATION_FAILED\n"
                  << infoLog << std::endl;
    }

    unsigned int shaderProgram = glCreateProgram();
    glAttachShader(shaderProgram, vertexShader);
    glAttachShader(shaderProgram, fragmentShader);
    glLinkProgram(shaderProgram);

    glGetProgramiv(shaderProgram, GL_LINK_STATUS, &success);
    if (!success)
    {
        glGetProgramInfoLog(shaderProgram, 512, NULL, infoLog);
        std::cerr << "ERROR::SHADER::PROGRAM::LINKING_FAILED\n"
                  << infoLog << std::endl;
    }

    glDeleteShader(vertexShader);
    glDeleteShader(fragmentShader);

    Particle *particles;
    int size = sizeof(Particle) * numParticles;
    particles = (Particle *)malloc(size);

    Particle *d_particles;
    hipMalloc((void **)&d_particles, size);

    float4 *d_colors;
    hipMalloc((void **)&d_colors, numColors * sizeof(float4));

    unsigned int seed = static_cast<unsigned int>(time(NULL));
    int threadsPerBlock = 256;
    int blocksPerGrid = (numParticles + threadsPerBlock - 1) / threadsPerBlock;

    initParticles<<<blocksPerGrid, threadsPerBlock>>>(d_particles, seed);
    initColors<<<1, numColors>>>(d_colors, seed);

    float attractionMatrix[numColors * numColors];
    for (int i = 0; i < numColors; i++)
    {
        for (int j = 0; j < numColors; j++)
        {
            attractionMatrix[i * numColors + j] = (i == j) ? 1.0f : -1.0f;
            // attractionMatrix[i * numColors + j] = -1.0f + static_cast<float>(rand()) / (static_cast<float>(RAND_MAX / (2.0f)));
        }
    }

    float *d_attractionMatrix;
    hipMalloc((void **)&d_attractionMatrix, numColors * numColors * sizeof(float));
    hipMemcpy(d_attractionMatrix, attractionMatrix, numColors * numColors * sizeof(float), hipMemcpyHostToDevice);

    unsigned int VBO, VAO;
    glGenVertexArrays(1, &VAO);
    glGenBuffers(1, &VBO);

    glBindVertexArray(VAO);

    glBindBuffer(GL_ARRAY_BUFFER, VBO);
    glBufferData(GL_ARRAY_BUFFER, size, particles, GL_DYNAMIC_DRAW);

    glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, sizeof(Particle), (void *)0);
    glEnableVertexAttribArray(0);

    glVertexAttribIPointer(1, 1, GL_INT, sizeof(Particle), (void *)(offsetof(Particle, color)));
    glEnableVertexAttribArray(1);

    glBindBuffer(GL_ARRAY_BUFFER, 0);

    glBindVertexArray(0);

    glUseProgram(shaderProgram);

    glPointSize(5.0f);

    float deltaTime = 0.0004f;
    float beta = 0.1f;
    float frictionFactor = 0.99f;

    int colorLoc = glGetUniformLocation(shaderProgram, "colors");

    while (!glfwWindowShouldClose(window))
    {
        glClear(GL_COLOR_BUFFER_BIT);

        updateParticles<<<blocksPerGrid, threadsPerBlock>>>(d_particles, d_attractionMatrix, numParticles, deltaTime, numColors, beta, frictionFactor);

        hipMemcpy(particles, d_particles, size, hipMemcpyDeviceToHost);

        glBindBuffer(GL_ARRAY_BUFFER, VBO);
        glBufferSubData(GL_ARRAY_BUFFER, 0, size, particles);

        float4 colors[numColors];
        hipMemcpy(colors, d_colors, numColors * sizeof(float4), hipMemcpyDeviceToHost);

        glUseProgram(shaderProgram);
        glUniform4fv(colorLoc, numColors, (float *)colors);

        glBindVertexArray(VAO);
        glDrawArrays(GL_POINTS, 0, numParticles);

        glfwSwapBuffers(window);
        glfwPollEvents();
    }

    glDeleteVertexArrays(1, &VAO);
    glDeleteBuffers(1, &VBO);

    hipFree(d_particles);
    hipFree(d_colors);
    hipFree(d_attractionMatrix);
    free(particles);

    glfwTerminate();
    return 0;
}
